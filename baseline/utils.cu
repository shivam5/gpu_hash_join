#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdint.h"
#include <cassert>
#include "helper.h"

// 32 bit Murmur3 hash
__device__ uint32_t hash(uint32_t k)
{
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (maxCapacity-1);
}

// Create a hash table. For linear probing, this is just an array of KeyValues
KeyValue* create_hashtable() 
{
    KeyValue* hashtable;
    hipMalloc(&hashtable, sizeof(KeyValue) * maxCapacity);

    // Initialize hash table to empty
    static_assert(kEmpty == 0xffffffff, "memset expected kEmpty=0xffffffff");
    hipMemset(hashtable, 0xff, sizeof(KeyValue) * maxCapacity);
    return hashtable;
}

// Insert the key/values in kvs into the hashtable
__global__ void gpu_hashtable_insert(KeyValue* hashtable, const KeyValue* kvs, unsigned int numkvs)
{
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadid < numkvs)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t value = kvs[threadid].value;
        uint32_t slot = hash(key);
        while (true)
        {
            uint32_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
            if (prev == kEmpty || prev == key)
            {
                hashtable[slot].value = value;
                return;
            }
            slot = (slot + 1) & (maxCapacity-1);
        }
    }
}
 
void insert_hashtable(KeyValue* pHashTable, const KeyValue* kvs, uint32_t num_kvs)
{
    // Copy the keyvalues to the GPU
    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int threadblocksize = std::min(1024, (int)num_kvs);
    int gridsize = (num_kvs + threadblocksize - 1) / threadblocksize;
    assert(gridsize * threadblocksize && "Number of threads are less");
    gpu_hashtable_insert<<<gridsize, threadblocksize>>>(pHashTable, device_kvs, num_kvs);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("GPU inserted %d items in %f ms (%f million keys/second)\n", num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

    hipFree(device_kvs);
}

__global__ void gpu_hashtable_print(KeyValue* deviceArr, const KeyValue* kvs, uint32_t tsize) {
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadid < tsize)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t value = kvs[threadid].value;
        uint32_t slot = hash(key);
        while (true)
        {
            if (deviceArr[slot].key != kEmpty) {
                if (key != deviceArr[slot].key) {
                    printf("Mismatch found: KV key: %d Hash key: %d\n", key, deviceArr[slot].key); 
                    return;
                }
                else if (value != deviceArr[slot].value) {
                    printf("Mismatch found: KV value: %u Hash value: %u\n", value, deviceArr[slot].value); 
                    return;
                }
                else {
                    printf("Match found: key: %d value: %u\n", deviceArr[slot].key, deviceArr[slot].value);
                    return;
                }
            }
            slot = (slot + 1) & (maxCapacity-1);
        }
    }
}

void print_arr_gpu(KeyValue* deviceArr, const KeyValue* kvs, uint32_t tsize) {
     // Copy the keyvalues to the GPU
    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * tsize);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * tsize, hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int threadblocksize = std::min(1024, (int)tsize);
    int gridsize = (tsize + threadblocksize - 1) / threadblocksize;

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    gpu_hashtable_print<<<gridsize, threadblocksize>>>(deviceArr, device_kvs, tsize);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("Printed GPU %d items in %f ms (%f million keys/second)\n", tsize, milliseconds, tsize / (double)seconds / 1000000.0f);

    hipFree(device_kvs);
}

__global__ void gpu_lookup(Match* rgpu, int* rid, KeyValue* t1, KeyValue* t2dev_kvs, uint32_t s1, uint32_t s2) {
    unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
    if (threadid < s2)
    {
        uint32_t k = t2dev_kvs[threadid].key;
        uint32_t v = t2dev_kvs[threadid].value;
        uint32_t slot = hash(k);
        while (true) {
            if (t1[slot].key == k) {
                uint32_t pos =  atomicAdd(rid, 1);
                rgpu[pos].key = k;
                rgpu[pos].value1 = t1[slot].value;
                rgpu[pos].value2 = v;
                return;
            }
            else if (t1[slot].key == kEmpty) {
                return;
            }
            slot = (slot + 1) & (maxCapacity-1);
        }
    }
}

int findMatches(Match* result, KeyValue* t1_hash, const KeyValue* t2_kvs, uint32_t s1, uint32_t s2) {
    Match* rgpu;
    hipMalloc(&rgpu, sizeof(Match) * s2);
    int* rid;
    hipMalloc(&rid, sizeof(int));
    hipMemset(rid, 0, sizeof(int));

    KeyValue* t2dev_kvs;
    hipMalloc(&t2dev_kvs, sizeof(KeyValue) * s2);
    hipMemcpy(t2dev_kvs, t2_kvs, sizeof(KeyValue) * s2, hipMemcpyHostToDevice);

    int threadblocksize = std::min(1024, (int)s2);
    int gridsize = (maxCapacity + threadblocksize - 1) / threadblocksize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    gpu_lookup<<<gridsize, threadblocksize>>>(rgpu, rid, t1_hash, t2dev_kvs, s1, s2);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }

    int rpos;
    hipMemcpy(&rpos, rid, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result, rgpu, sizeof(Match) * rpos, hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("Found %d matches in %f ms (%f million keys/second)\n", rpos, milliseconds, rpos / (double)seconds / 1000000.0f);
    
    return rpos;
}