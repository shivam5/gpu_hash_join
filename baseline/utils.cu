#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdint.h"
#include "helper.h"
#include "vector"

// 32 bit Murmur3 hash
__device__ uint32_t hash(uint32_t k)
{
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (maxCapacity-1);
}

// Create a hash table. For linear probing, this is just an array of KeyValues
KeyValue* create_hashtable() 
{
    // Allocate memory
    KeyValue* hashtable;
    hipMalloc(&hashtable, sizeof(KeyValue) * maxCapacity);

    // Initialize hash table to empty
    static_assert(kEmpty == 0xffffffff, "memset expected kEmpty=0xffffffff");
    hipMemset(hashtable, 0xff, sizeof(KeyValue) * maxCapacity);

    return hashtable;
}

// Insert the key/values in kvs into the hashtable
__global__ void gpu_hashtable_insert(KeyValue* hashtable, const KeyValue* kvs, unsigned int numkvs)
{
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadid < numkvs)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t value = kvs[threadid].value;
        uint32_t slot = hash(key);
        printf("Inserting pos: %u key: %d val: %d\n", slot, key, value);
        while (true)
        {
            uint32_t prev = atomicCAS(&hashtable[slot].key, kEmpty, key);
            if (prev == kEmpty || prev == key)
            {
                hashtable[slot].value = value;
                return;
            }

            slot = (slot + 1) & (maxCapacity-1);
        }
    }
}
 
void insert_hashtable(KeyValue* pHashTable, const KeyValue* kvs, uint32_t num_kvs)
{
    // Copy the keyvalues to the GPU
    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Insert all the keys into the hash table
    int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
    gpu_hashtable_insert<<<gridsize, threadblocksize>>>(pHashTable, device_kvs, (uint32_t)num_kvs);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    // printf("GPU inserted %d items in %f ms (%f million keys/second)\n", num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

    hipFree(device_kvs);
}

void transfer_data(KeyValue* hostArr, KeyValue* deviceArr, uint32_t tsize) {
    hipMemcpy(hostArr, deviceArr, sizeof(KeyValue) * tsize, hipMemcpyDeviceToHost);
}

__global__ void gpu_lookup(Match* rgpu, int* rid, KeyValue* t1, KeyValue* t2dev_kvs, uint32_t s1, uint32_t s2) {
    unsigned int threadid = blockDim.x * blockIdx.x + threadIdx.x;
    if (threadid < s2)
    {
        uint32_t k = t2dev_kvs[threadid].key;
        uint32_t v = t2dev_kvs[threadid].value;
        uint32_t hval = hash(k);
        if (t1[hval].key == k) {
            uint32_t pos =  atomicAdd(rid, 1);
            rgpu[pos].key = k;
            rgpu[pos].value1 = t1[hval].value;
            rgpu[pos].value2 = v;
        }
    }
}

int findMatches(Match* result, KeyValue* t1_hash, const KeyValue* t2_kvs, uint32_t s1, uint32_t s2) {
    Match* rgpu;
    hipMalloc(&rgpu, sizeof(Match) * s2);
    int* rid;
    hipMalloc(&rid, sizeof(int));
    hipMemset(rid, 0, sizeof(int));

    KeyValue* t2dev_kvs;
    hipMalloc(&t2dev_kvs, sizeof(KeyValue) * s2);
    hipMemcpy(t2dev_kvs, t2_kvs, sizeof(KeyValue) * s2, hipMemcpyHostToDevice);

    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_lookup, 0, 0);
    
    int gridsize = (maxCapacity + threadblocksize - 1) / threadblocksize;
    gpu_lookup<<<gridsize, threadblocksize>>>(rgpu, rid, t1_hash, t2dev_kvs, s1, s2);
    hipDeviceSynchronize();

    int rpos;
    hipMemcpy(&rpos, rid, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(result, rgpu, sizeof(Match) * rpos, hipMemcpyDeviceToHost);
    return rpos;
}